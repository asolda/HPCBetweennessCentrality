#include <iostream>
#include <stack>
#include <list>
#include <queue>
#include <string>
#include <stdio.h>
#include <string.h>
#include <algorithm>

#include "mpi.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuBCStruct.h"
#include "constant.h"
#include "graph_indexed.h"

#include "BC_gpu.cu"
#include "BC_cpu.cpp"

#define DEVICE_COUNT_FAIL 8

#define MASTER 0

#define FROM_MASTER 2
#define TO_MASTER 3

int main(int argc, char **argv) {

	int i;
	int rank, size;
	//processor name
	char *p_name;
	int name_length;
    int handshakeBit;
    std::vector<int> masters;
	MPI_Status status;
	//store available GPUs for each node
	int devCount;
	int totalCount;
    int recvTime, maxTime;

    int unit;

    double begintime, endtime;

    //key: proc name, value: master && numdev
    std::map<std::string, std::pair<int, int> > fullMap;
    //key: processor name, value: process rank
    std::map<std::string, int> procMap;
    //key: processor name, value: master_id && slave_ids vector
    std::map<std::string, std::pair<int, std::vector<int> > > subMasterMap;
	
	MPI_Init(&argc, &argv);

    int CPU_VALUE = atoi(argv[2]);
    int GPU_VALUE = 10 - CPU_VALUE;

	MPI_Comm_size(MPI_COMM_WORLD, &size); 
	//size = MPI::COMM_WORLD.Get_size();
	MPI_Comm_rank(MPI_COMM_WORLD, &rank); 
    //rank = MPI::COMM_WORLD.Get_rank();

    p_name = (char*) malloc(MPI_MAX_PROCESSOR_NAME * sizeof(char));

    if(rank != MASTER) { //SLAVE
    	MPI_Get_processor_name(p_name, &name_length);
    	
        MPI_Send(p_name, name_length, MPI_CHAR, MASTER, TO_MASTER, MPI_COMM_WORLD);

        MPI_Recv(&handshakeBit, 1, MPI_INT, MASTER, FROM_MASTER, MPI_COMM_WORLD, &status);
        if(handshakeBit == 1) {
            //I am processor's master
            hipGetDeviceCount(&devCount);
            MPI_Send(&devCount, 1, MPI_INT, MASTER, TO_MASTER, MPI_COMM_WORLD);            
        }
    } else { //MASTER 
        begintime = MPI::Wtime();
    
        std::pair<std::string, std::pair<int, std::vector<int> > > nextEntry;
        std::pair<std::string, int> toInsert;
    	for(i = 1; i < size; i++) {
    		MPI_Recv(p_name, MPI_MAX_PROCESSOR_NAME, MPI_CHAR, i, TO_MASTER, MPI_COMM_WORLD, &status);
            std::string str(p_name);
            //map master -> slaves
            if(subMasterMap.find(str) != subMasterMap.end()) {
                subMasterMap[str].second.push_back(i);
            } else {
                nextEntry.first = str;
                nextEntry.second.first = i;
                subMasterMap.insert(nextEntry);
            }
            toInsert.first = str;
            toInsert.second = i;
            procMap.insert(toInsert); 
    	}
        
        std::map<std::string, int>::iterator iter;
        for(iter = procMap.begin(); iter != procMap.end(); ++iter) {
            //std::cout << "Processor name: " << iter->first << " Processor master: " << iter->second << "\n";
            masters.push_back(iter->second);
        }
        for(i = 1; i < size; i++) {
            //sending 1 to masters and 0 to non-masters
            if(find(masters.begin(), masters.end(), i) != masters.end()){
                //masters found
                handshakeBit = 1;
                MPI_Send(&handshakeBit, 1, MPI_INT, i, FROM_MASTER, MPI_COMM_WORLD);
            } else {
                handshakeBit = 0;
                MPI_Send(&handshakeBit, 1, MPI_INT, i, FROM_MASTER, MPI_COMM_WORLD);
            }
        }

        //receiving numbers of cuda devices
        std::pair<std::string, std::pair<int, int> > newToInsert;
        std::pair<int, int> nextValue;
        totalCount = 0;
        for(std::vector<int>::iterator it = masters.begin(); it != masters.end(); ++it) {
            MPI_Recv(&devCount, 1, MPI_INT, *it, TO_MASTER, MPI_COMM_WORLD, &status);
            if(devCount >= DEVICE_COUNT_FAIL || devCount < 0) devCount = 0;
            totalCount += devCount;
            for(iter = procMap.begin(); iter != procMap.end(); ++iter) {
                if(*it == iter->second) {
                    //std::cout << "Processor name: " << iter->first << " Processor master: " << *it << " Number of cuda devices: " << devCount << "\n";
                    newToInsert.first = iter->first;
                    nextValue.first = *it;
                    nextValue.second = devCount;
                    newToInsert.second = nextValue;
                    fullMap.insert(newToInsert);
                }
                continue;
            }
            
        }

        //computing minimum input size
        unit = (size * CPU_VALUE) + (totalCount * GPU_VALUE);

        //printing info
        /*
        std::cout << "CPU number = " << size << " CPU value = " << CPU_VALUE << "\nGPU number = " << totalCount << " GPU value = " << GPU_VALUE<< "\nUnit value: " << unit << "\n";

        std::cout << "\nMore info about cluster capacities:\n\n";
        std::map<std::string, std::pair<int, int> >::iterator info;
        for(info = fullMap.begin(); info != fullMap.end(); ++info) {        
            std::cout << "Node: " << info->first << " Node master: " << info->second.first << "\n";
            std::cout << "Available CPUs: " << size / fullMap.size() << " Available GPUs: " << info->second.second << "\n";
            std::cout << "Processes on this node: ";
            std::cout << "(" << subMasterMap[info->first].first << ") ";
            std::vector<int> slaves = subMasterMap[info->first].second;
            for(i = 0; i < slaves.size(); i++) {
                std::cout << slaves[i] << " ";
            }
            std::cout << "\n\n";
        }
        */
    }//FINE MASTER

    char *filename = argv[1];

    //caricamento grafo da file
    GraphIndexed* pGraph = new GraphIndexed();
    if(!pGraph->Load(filename)) {
        std::cout << "Error while loading file" << std::endl;
        return -1;
    }

    int n = pGraph->NumberOfNodes();

    if(rank == MASTER) {
        int slice = n / unit;
        int currentPos = 0;

        //use GPU, start, end
        int toSend[3] = {0, 0, 0};

        //work distribution
        int availGPUs = 0;
        handshakeBit = 0;
        std::map<std::string, std::pair<int, int> >::iterator info;
        //for each processor
        for(info = fullMap.begin(); info != fullMap.end(); ++info) { 
            std::vector<int> slaves = subMasterMap[info->first].second;
            availGPUs = info->second.second;
            //for each slave on proc
            for(i = 0; i < slaves.size(); i++) { 
                if(availGPUs > 0 && GPU_VALUE > 0) {
                    //use GPU
                    toSend[0] = 1;
                    toSend[1] = currentPos;
                    currentPos += ((slice) * GPU_VALUE) + 1;
                    toSend[2] = currentPos;
                    availGPUs--;
                } else {
                    toSend[0] = 0;
                    toSend[1] = currentPos;
                    currentPos += ((slice) * CPU_VALUE) + 1;
                    toSend[2] = currentPos;
                }      
                MPI_Send(toSend, 3, MPI_INT, slaves[i], FROM_MASTER, MPI_COMM_WORLD);

            }
            //send to master
            MPI_Send(toSend, 3, MPI_INT, subMasterMap[info->first].first, FROM_MASTER, MPI_COMM_WORLD);
        }
    }

    //inizializzazione CPU
    cuGraph* pCUGraph = NULL;
    cuBC*    pBCData  = NULL;
    initGraph(pGraph, pCUGraph);
    initBC(pCUGraph, pBCData);

    if(rank != MASTER) {
        int recv[3];

        MPI_Recv(recv, 3, MPI_INT, MASTER, FROM_MASTER, MPI_COMM_WORLD, &status);
        if(recv[0] == 1) {
            //inizializzazione GPU
            cuGraph* pGPUCUGraph = NULL;
            cuBC*    pGPUBCData  = NULL;
            initGPUGraph(pCUGraph, pGPUCUGraph);
            initGPUBC(pBCData, pGPUBCData);

            //computing BC on GPU
            gpuComputeBCOpt(pGPUCUGraph, pGPUBCData, recv[1], recv[2]);
            //copy results
            copyBackGPUBC(pGPUBCData, pBCData);
            //free memory
            freeGPUGraph(pGPUCUGraph);
            freeGPUBC(pGPUBCData);
        } else {
            //compute BC on CPU
            cpuComputeBCOpt(pCUGraph, pBCData, recv[1], recv[2]);
        }
    }

    float *BCs = (float*)calloc(n, sizeof(float));

    MPI_Reduce(pBCData->nodeBC, BCs, n, MPI_FLOAT, MPI_SUM, MASTER, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);

    if(rank == MASTER) {
        endtime = MPI::Wtime();
        std::cout << "Time: " << endtime - begintime << " ms Graph: " << filename << " alpha: " << CPU_VALUE << std::endl;
    }

    freeGraph(pCUGraph);
    freeBC(pBCData);
    delete pGraph;  

    MPI_Finalize(); 
}